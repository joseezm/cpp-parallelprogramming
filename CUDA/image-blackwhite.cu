#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;


__global__
void cudaGrayScale(float *R, float *G, float *B, float* gray, int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < n) {
        gray[i] = static_cast<float>((R[i] * 0.21 + G[i] * 0.71 + B[i] * 0.07) / 350.0);
    }
}

void grayscale(float* R, float* G, float* B, float* grayscale, int n){
    int size = n * sizeof(float);
    float *d_R, *d_G, *d_B, *d_gray;
    hipMalloc((void **) &d_R, size);
    hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_G, size);
    hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_gray, size);

    cudaGrayScale<<<ceil(n/1024.0), 1024>>>(d_R, d_G, d_B, d_gray, n);
    hipMemcpy(grayscale, d_gray, size, hipMemcpyDeviceToHost);

    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);
    hipFree(d_gray);
}


int main(){
    string image_path = "ejemplo.jpg";
    Mat img = imread(image_path, IMREAD_COLOR);
    
    int filas = img.rows;
    int columnas = img.cols;

    float * R = new float[filas*columnas];
    float * G = new float[filas*columnas];
    float * B = new float[filas*columnas];

    float* GrayScaleMatrix = new float[filas * columnas];

    for(int i = 0; i < filas; ++i){
        for(int j = 0; j < columnas; ++j){
            int index = columnas * i + j;
            R[index] = (float)img.at<cv::Vec3b>(i, j)[2];
            G[index] = (float)img.at<cv::Vec3b>(i, j)[1];
            B[index] = (float)img.at<cv::Vec3b>(i, j)[0];
        }
    }

    grayscale(R, G, B, GrayScaleMatrix, columnas * filas);
    Mat gray = Mat(filas, columnas, CV_32FC1, GrayScaleMatrix);
    gray.convertTo(gray, CV_8UC3, 255.0);
    imwrite("./ejemplobyn.jpg", gray);


    return 0;
}