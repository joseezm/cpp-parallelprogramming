#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLUR_SIZE 1

using namespace std;
using namespace cv;


__global__
void blurKernel (unsigned char * in, unsigned char * out, int w, int h){
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    

    if(Col < w && Row < h){
        int pixVal = 0;
        int pixels = 0;

        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow){
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol){
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;
                if(curRow > -1 && curRow < h && curCol > -1 && curCol < w){
                    pixVal += in[curRow * w + curCol];
                    pixels++;
                }
            }
        }
        out[Row * w + Col] = (float)(pixVal/pixels);
    }

}



void blur_image(unsigned char * Input_image, int Height, int Width){
    unsigned char * Dev_Input_Image = NULL;
    unsigned char * Dev_Output_Image = NULL;
    int n = Height * Width;

    hipMalloc((void**)&Dev_Input_Image, Height * Width);
    hipMemcpy(Dev_Input_Image, Input_image, Height * Width, hipMemcpyHostToDevice);

    hipMalloc((void**)&Dev_Output_Image, Height * Width);

    dim3 Grid_Image(Width,Height);
    blurKernel << <Grid_Image,1024>> >  (Dev_Input_Image, Dev_Output_Image, Width, Height);

    hipMemcpy(Input_image, Dev_Output_Image, Width * Height, hipMemcpyDeviceToHost);
    hipFree(Dev_Input_Image);
    hipFree(Dev_Input_Image);

}

int main(){
    string image_path = "goku.jpg";
    Mat img = imread(image_path, IMREAD_COLOR);

    blur_image(img.data, img.rows, img.cols);

    imwrite("blur_image_res.jpg", img);


    return 0;
}