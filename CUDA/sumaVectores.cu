#include <iostream>
#include <stdlib.h>
#include <ctime>

#include <hip/hip_runtime.h>


using namespace std;


__global__
void vecAddKernel(float * A, float *B, float *C, int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i<n) C[i] = A[i] + B[i];

}


void vecAdd(float * A, float *B, float *C, int n){
	int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_B, size);
    hipMemcpy(d_B,B,size, hipMemcpyHostToDevice);

    hipMalloc((void**) &d_C, size);

    dim3 dimGrid(ceil(n/256.0),1,1);
    dim3 dimBlock(256,1,1);
    vecAddKernel<<<dimGrid,dimBlock >>> (d_A, d_B, d_C, n);

    hipMemcpy(C,d_C,size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

int main(){
	
    float * A = new float[1000];
    float * B = new float[1000];
    float * C = new float[1000];

    for(int i = 0; i<1000; i++){
        A[i] = i;
        B[i] = i;
        C[i] = 1;
    }

    vecAdd(A,B,C,1000);

    for(int i = 0; i<20; i++)
        cout << C[i] << endl;

	return 0;
}