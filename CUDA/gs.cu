#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLUR_SIZE 1

using namespace std;
using namespace cv;


__global__ void blanco_negro(unsigned char* Image, int Channels){
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++){
        if(i==0){
            Image[idx + i] = Image[idx + i] * 0.07f/350;
        }
        else if(i==1){
            Image[idx + i] = Image[idx + i] * 0.71f/350;
        }
        else if(i==2){
            Image[idx + i] = Image[idx + i] * 0.21f/350;
        }
		
	}
}

void Image_blanco_negro(unsigned char* Input_Image, int Height, int Width, int Channels){
	unsigned char* Dev_Input_Image = NULL;

	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	blanco_negro << <Grid_Image, 1024 >> >(Dev_Input_Image, Channels);

	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}



int main(){
	Mat Input_Image = imread("goku.jpg");

	cout << "Height: " << Input_Image.rows << ", Width: " << Input_Image.rows << ", Channels: " << Input_Image.channels() << endl;

	Image_blanco_negro(Input_Image.data, Input_Image.rows, Input_Image.rows, Input_Image.channels());

	imwrite("blancoynegro.jpg", Input_Image);
	system("pause");
	return 0;
}